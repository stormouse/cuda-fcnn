#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "functions.cuh"
#include "nn.cuh"

#include <vector>
#include <random>

std::vector<float> generateRandomVector(size_t size) {
    std::vector<float> v(size);

    // Random number generator
    std::random_device rd;  // Obtain a random number from hardware
    std::mt19937 gen(rd()); // Seed the generator

    // Define the range for uniform distribution
    std::uniform_real_distribution<> dis(0.0, 1.0);  // Range [0.0, 1.0)

    // Populate the vector with random numbers
    for (auto& elem : v) {
        elem = dis(gen) >= 0.5f ? 1.0f : 0.0f;  // Each call to dis(gen) generates a new random float
    }

    return v;
}

int main() {
    float *d_input;
    int dim = 16;
    CUDA_CALL(hipMalloc((void **)&d_input, dim * sizeof(float)));

    std::vector<int> layers{dim, 4, dim};
    Model model(layers);
    float lr = 0.01f;

    auto h_input = generateRandomVector(dim);

    for (int t = 0; t < 100; t++)
    {
        float epochLoss = 0.0f;
        for (int i = 0; i < 1000; i++)
        {
            h_input = generateRandomVector(dim);
            CUDA_CALL(hipMemcpy(d_input, h_input.data(), dim * sizeof(float), hipMemcpyHostToDevice));

            auto loss = model.fit(d_input, d_input, lr);
            epochLoss += loss;
        }
        std::cout << "epoch " << t << ": loss = " << epochLoss / 1000 << ", lr = " << lr << "\n";
        lr *= 0.98f;
    }

    for (int i = 0; i < layers.size() - 1; i++) {
        std::cout << "W:" << std::endl;
        printKernel<<<1, 1>>>(model.layers[i]->W, layers[i] * layers[i + 1]);
        hipDeviceSynchronize();
        std::cout << "b:" << std::endl;
        printKernel<<<1, 1>>>(model.layers[i]->b, layers[i + 1]);
        hipDeviceSynchronize();
        std::cout << "a:" << std::endl;
        printKernel<<<1, 1>>>(model.layers[i]->a, layers[i + 1]);
        hipDeviceSynchronize();
        std::cout << "\n\n\n" << std::endl;
    }

    printKernel<<<1, 1>>>(model.layers.back()->a, layers.back());
    hipDeviceSynchronize();
    for (int i = 0; i < layers.back(); i++) {
        std::cout << "Label[" << i << "] = " << h_input[i] << "\n";
    }

    std::cout << "\n";

    hipDeviceSynchronize();

    CUDA_CALL(hipFree(d_input));

    return 0;
}